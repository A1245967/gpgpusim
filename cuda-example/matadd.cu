#include<stdio.h>
#include<hip/hip_runtime.h>


#define N 512

__global__ void MatAdd(float *A, float *B, float *C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // C = A + B
    if(i < N && j < N){
        C[i*N+j] = A[i*N+j] + B[i*N+j];
    }

}

int main()
{

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    int i;

    // allocate the host memory
    h_A = (float*)malloc(N * N * sizeof(float));
    h_B = (float*)malloc(N * N * sizeof(float));
    h_C = (float*)malloc(N * N * sizeof(float));

    // init host data
    for(i = 0; i < (N * N); i++)
    {
	    h_A[i] = 1.0;
	    h_B[i] = 2.0;
	    h_C[i] = 0.0;
    }

    // allocate device memory
    hipMalloc((void**)&d_A,  N * N * sizeof(float));
    hipMalloc((void**)&d_B,  N * N * sizeof(float));
    hipMalloc((void**)&d_C,  N * N * sizeof(float));
    
    // transfer host data to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // fill in correct thread/blocks
    unsigned int x_size = 32;
    unsigned int y_size = 32;
    unsigned int grid_rows = (N + x_size - 1) / x_size;
    unsigned int grid_cols = (N + y_size - 1) / y_size;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(x_size, y_size);

    // Launch MatAdd kernel
    MatAdd<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // transfer device data back to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    int all_ok = 1;
    for(i = 0; i < (N * N); i++)
    {
        if(h_C[i]!= 3.0)
        {
	        all_ok = 0;
	    }
    }

    if(all_ok)
    {
        printf("all results are correct!!!\n");
    }
    else
    {
        printf("incorrect results\n");
    }

    // free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
